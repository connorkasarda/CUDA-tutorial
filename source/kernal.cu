#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "kernal.h"

__global__ void AddIntsCUDA(int* a, int* b)
{
    a[0] += b[0];
}

void RunMyFirstKernal()
{
    std::cout << "Running first kernal example..." << std::endl;

    int a = 5, b = 9;
    
    // These are device pointers, hence the d_ prefix
    int *d_a, *d_b;

    // Typically want to detect if these are successful
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    
    AddIntsCUDA<<<1, 1>>>(d_a, d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout << "The answer is " << a << std::endl;

    // Make sure to free memory that was allocated
    hipFree(d_a);
    hipFree(d_b);

    std::cout << std::endl;
}
