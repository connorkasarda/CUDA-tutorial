#include <iostream>
#include <hip/hip_runtime.h>

__global__ void AddIntsCUDA(int* a, int* b)
{
    a[0] += b[0];
}

int main(int argc, char ** argv)
{
    int a = 5, b = 9;
    
    // These are device pointers, hence the d_ prefix
    int *d_a, *d_b;

    // Typically want to detect if these are successful
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    
    AddIntsCUDA<<<1, 1>>>(d_a, d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout << "The answer is " << a << std::endl;

    // Make sure to free memory that was allocated
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
